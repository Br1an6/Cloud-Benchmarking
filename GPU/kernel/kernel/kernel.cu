
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
__global__ void
floatAdd(const float *A, const float *B, float *C, int numElements)
{
    //Get the index 
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}


int main(void)
{


    //Define number of elements
    int numElements = 50000;

	//FLOPS
    size_t size = numElements * sizeof(float);
    printf("FLoatoperations on %d elements\n", numElements);

    // Allocate the host input vector A
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

 
    // Initialize the host input with random generators
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }

    // Allocate the device input vector A
    float *d_A = NULL;
    float *d_B = NULL;
    float *d_C = NULL;

    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);
	    
    // Copy the host input data in host memory to the device input variables in device memory
    printf("Copy input data from the host memory to device memory\n");
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Launch the  FloatAdd CUDA Kernel
    int threadsPerBlock = 3584;  //probably more than that
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

	//Time to calculate
	//Timing
	clock_t start, stop;

    start = clock();
	printf("%6.3f\n", (double)start);

    floatAdd<<<1, 1>>>(d_A, d_B, d_C, numElements);
     

    // Copy the device result vector in device memory to the host result vector in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    
	//End time
	//check end time
	 stop = clock();
				  
	//10^9 for G for flops,iops,hops,qops
	double elapsed = double(stop-start);

	printf("Time taken : %6.3f\n", double(stop-start));
	printf  ("Time taken in GFLOPS: %9.9f\n", elapsed/pow(10,9));

	// Clearing device global memory
     hipFree(d_A);
	 hipFree(d_B);
	 hipFree(d_C);

    // Clearing host memory
    free(h_A);
    free(h_B);
    free(h_C);

    printf("Process completed\n");
    return 0;
}

