#include "hip/hip_runtime.h"
#include <cstdio>
#include <ctime>
#include <cmath> 
#include <pthread.h>

  __global__ void doubleKernel(double * x, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
      x[i] = x[i] + 73.333;
	  x[i] = x[i] *12.22;
    //  printf("Value of x[i] is %0.2f\n", x[i]);
    }
  }

__global__ void intKernel(int * x, int n) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
    x[i] = x[i] + 25635;
	x[i] = x[i] * 35;
   // printf("Value of x[i] is %d\n",  x[i]);
  }
}

__global__ void shortKernel(short int * x, int n) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
    x[i] = x[i] + 52;
	x[i] = x[i] * 9;
   // printf("Value of x[i] is %d\n", x[i]);
  }
}

__global__ void charKernel(char * x, int n) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  for (int i = tid; i < n; i += blockDim.x * gridDim.x) {

    x[i] = 'h';
	if (x[i]=='h'){
	  char t = 'i';
	  //printf("Test char is %c\n", t);
	  	}
   // printf("Value of x[i] is %c\n", x[i]);
  }
}

int main(int argc, char * * argv) {
  //Define number of elements
	const int numElements = 50000;

  //Timing
  clock_t start, stop;
  double elapsed;

  //define number of streams - 2496
  const int cores = 2496;
  hipStream_t streams[cores];

  int op = 0;

  if (argc < 1) {
    printf("Please enter operation type");
    return 0;
  }

  if (argc > 0) {
    op = atoi(argv[1]);
    printf("Optn is %d\n", op);
  }

  switch (op) {

  case 1:
    printf("You selected FLOPS\n");
	 double *flops[cores];

	
    start = clock();

    for (int i = 0; i < cores; i++) {
      hipStreamCreate( & streams[i]);
      hipMalloc( & flops[i], cores * sizeof(double));

      // launch one worker kernel per stream
      doubleKernel <<< 1, 64, 0, streams[i] >>> (flops[i], numElements);
      printf("Creating kernel for flops  %d\n", i);

    }
    //End time
    stop = clock();
    elapsed = double(stop - start);
    printf("Time taken in GFLOPS: %9.9f\n", elapsed / pow(10, 9));
    break;

  case 2:
    printf("You selected IOPS\n");
    int * iops[cores];

    start = clock();

    for (int i = 0; i < cores; i++) {
      hipStreamCreate( & streams[i]);
      hipMalloc( & iops[i], cores * sizeof(int));

      // launch one worker kernel per stream
      intKernel << < 1, 64, 0, streams[i] >>> (iops[i], numElements);
      printf("Creating kernel for iops  %d\n", i);

    }
    //End time
    stop = clock();
    elapsed = double(stop - start);
    printf("Time taken in GIOPS: %9.9f\n", elapsed / pow(10, 9));
    break;
  case 3:
    printf("You selected HOPS\n");
    short int * hops[cores];

    start = clock();

    for (int i = 0; i < cores; i++) {
      hipStreamCreate( & streams[i]);
      hipMalloc( & hops[i], cores * sizeof(short int));

      // launch one worker kernel per stream
      shortKernel << < 1, 64, 0, streams[i] >>> (hops[i], numElements);
      printf("Creating kernel for hops  %d\n", i);

    }
    //End time
    stop = clock();
    elapsed = double(stop - start);
    printf("Time taken in GHOPS: %9.9f\n", elapsed / pow(10, 9));
    break;
  case 4:
    printf("You selected QOPS\n");
    char * qops[cores];

     start = clock();

    for (int i = 0; i < cores; i++) {
      hipStreamCreate( & streams[i]);
      hipMalloc( & qops[i], cores * sizeof(char));

      // launch one worker kernel per stream
      charKernel << < 1, 64, 0, streams[i] >>> (qops[i], numElements);
      printf("Creating kernel for qops  %d\n", i);

    }
    //End time
    stop = clock();
    elapsed = double(stop - start);
    printf("Time taken in GQOPS: %9.9f\n", elapsed / pow(10, 9));
    break;

  default:
    printf("Please enter valid operation type 1 or 2 or 3 or 4 for flops/iops/hops/qops respectively.\n");
    break;
  }

  printf("Process completed\n");
  hipDeviceReset();

  return 0;
}
