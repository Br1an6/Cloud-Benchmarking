#include "hip/hip_runtime.h"
#include < stdio.h >
#include < time.h >
#include < math.h > 
#include < pthread.h >

  __global__ void doubleKernel(double * x, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
      // x[i] = sqrt(pow(3.14159,i));
      x[i] = x[i] * 2.333 + x[i] * 3.444;
      printf("Value of x[i] is %0.2d\n", &x[i]);
    }
  }

__global__ void intKernel(int * x, int n) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
    x[i] = x[i] * 2 + x[i] * 3;
    printf("Value of x[i] is %d\n", & x[i]);
  }
}

__global__ void shortKernel(short int * x, int n) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
    x[i] = x[i] * 2 + x[i] * 3;
    printf("Value of x[i] is %d\n", x[i]);
  }
}

__global__ void charKernel(char * x, int n) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  for (int i = tid; i < n; i += blockDim.x * gridDim.x) {

    printf("Value of x[i] is %s\n", x[i]);
  }
}

int main(int argc, char * * argv) {
  //Define number of elements
	const int numElements = 50000;

  printf("Char is %zd: \n", sizeof(char));

  //Timing
  clock_t start, stop;
  double elapsed;

  //define number of streams 
  const int cores = 8;
  hipStream_t streams[cores];

  int op = 0;

  if (argc < 1) {
    printf("Please enter operation type");
    return 0;
  }

  if (argc > 0) {
    op = atoi(argv[1]);
    printf("Optn is %d\n", op);
  }

  switch (op) {

  case 1:
    printf("You selected FLOPS\n");
	 int *flops[cores];

    start = clock();

    for (int i = 0; i < cores; i++) {
      hipStreamCreate( & streams[i]);
      hipMalloc( & flops[i], cores * sizeof(int));

      // launch one worker kernel per stream
      intKernel << < 1, 32, 0, streams[i] >>> (flops[i], numElements);
      printf("Creating kernel for flops  %d\n", i);

    }
    //End time
    //check end time
    stop = clock();
    //10^9 for G for flops,iops,hops,qops
    elapsed = double(stop - start);
    printf("Time taken : %6.3f\n", double(stop - start));
    printf("Time taken in GFLOPS: %9.9f\n", elapsed / pow(10, 9));
    break;

  case 2:
    printf("You selected IOPS\n");
    int * iops[cores];

    start = clock();

    for (int i = 0; i < cores; i++) {
      hipStreamCreate( & streams[i]);
      hipMalloc( & iops[i], cores * sizeof(int));

      // launch one worker kernel per stream
      intKernel << < 1, 32, 0, streams[i] >>> (iops[i], numElements);
      printf("Creating kernel for iops  %d\n", i);

    }
    //End time
    //check end time
    stop = clock();
    //10^9 for G for flops,iops,hops,qops
    elapsed = double(stop - start);
    printf("Time taken : %6.3f\n", double(stop - start));
    printf("Time taken in GIOPS: %9.9f\n", elapsed / pow(10, 9));
    break;
  case 3:
    printf("You selected HOPS\n");
    short int * hops[cores];

    start = clock();

    for (int i = 0; i < cores; i++) {
      hipStreamCreate( & streams[i]);
      hipMalloc( & hops[i], cores * sizeof(short int));

      // launch one worker kernel per stream
      shortKernel << < 1, 16, 0, streams[i] >>> (hops[i], numElements);
      printf("Creating kernel for iops  %d\n", i);

    }
    //End time
    //check end time
    stop = clock();
    //10^9 for G for flops,iops,hops,qops
    elapsed = double(stop - start);
    printf("Time taken : %6.3f\n", double(stop - start));
    printf("Time taken in GHOPS: %9.9f\n", elapsed / pow(10, 9));
    break;
  case 4:
    printf("You selected QOPS\n");
    char * qops[cores];

    for (int j = 0; j < cores; j++) {
      qops[j] = "a";
    }
    for (int k = 0; k < cores; k++) {
      printf("q is %s\n", qops[k]);
    }

    start = clock();

    for (int i = 0; i < cores; i++) {
      hipStreamCreate( & streams[i]);
      hipMalloc( & qops[i], cores * sizeof(char));

      // launch one worker kernel per stream
      charKernel << < 1, 8, 0, streams[i] >>> (qops[i], numElements);
      printf("Creating kernel for qops  %d\n", i);

    }
    //End time
    //check end time
    stop = clock();
    //10^9 for G for flops,iops,hops,qops
    elapsed = double(stop - start);
    printf("Time taken : %6.3f\n", double(stop - start));
    printf("Time taken in GQOPS: %9.9f\n", elapsed / pow(10, 9));
    break;

  default:
    printf("Please enter valid operation type 1 or 2 or 3 or 4 for flops/iops/hops/qops respectively.\n");
    break;
  }

  printf("Process completed\n");
  hipDeviceReset();

  return 0;
}